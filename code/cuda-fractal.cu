#include "hip/hip_runtime.h"

  
#include <stdio.h>
#include <unistd.h>
#include <err.h>
#include <stdint.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


void checkErr(hipError_t err, char* msg)
{
    if (err != hipSuccess){
        fprintf(stderr, "%s (error code %d: '%s'", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__device__ int compute_point( double x, double y, int max )
{
	double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;

    int iter;

    for (iter = 0; iter < max; iter++){
		zi = zr * zi;
		zi += zi;
		zi += y;
		zr = (zrsqr - zisqr) + x;
		zrsqr = zr * zr;
		zisqr = zi * zi;
		
		if (zrsqr + zisqr >= 4.0) break;
    }
	
    return iter;
}

__global__ void compute_image_kernel(double xmin, double xmax, double ymin, double ymax, int maxiter, int width, int height, char* result, int num_of_chanels) 
{
    

    int pix_per_thread = (width * height*num_of_chanels) / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    int iter;
	double xstep = (xmax-xmin) / (width-1);
	double ystep = (ymax-ymin) / (height-1);

    for (int i = offset; i < offset + pix_per_thread; i++){
        
        int iw = i%width;
        int ih = i/height;
        
        double x = xmin + iw*xstep;
		double y = ymin + ih*ystep;

        iter = compute_point(x, y, maxiter);
        if(num_of_chanels>1){
            result[num_of_chanels*(ih * width + iw)]  = (char) ((int)(iter * iter/maxiter)%255);
            result[num_of_chanels*(ih * width + iw) + 1]  = (char)(iter%256); 
            result[num_of_chanels*(ih * width + iw) + 2]  = (char) ((iter*iter)%255);    
        }else{
		    int gray = 255 * iter / maxiter;
            result[ih * width + iw]  = (char)gray;
        }
      


    
    }

    if (gridDim.x * blockDim.x * pix_per_thread < width * height && tId < (width * height) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        
        int iw = i%width;
        int ih = i/height;
        
        double x = xmin + iw*xstep;
		double y = ymin + ih*ystep;

        iter = compute_point(x, y, maxiter);
        if(num_of_chanels>1){
            result[num_of_chanels*(ih * width + iw)]  = (char) ((int)(iter * iter/maxiter)%255);
            result[num_of_chanels*(ih * width + iw) + 1]  = (char)(iter%256); 
            result[num_of_chanels*(ih * width + iw) + 2]  = (char) ((iter*iter)%255);    
        }else{
		    int gray = 255 * iter / maxiter;
            result[ih * width + iw]  = (char)gray;
        }

    }
    
    
}

__host__  static void run(double xmin, double xmax, double ymin, double ymax, int width, int height, int max_iter, char* result, int rgb)

{   

    int num_of_chanels;

    if(rgb==1){
        num_of_chanels = 3;

    }else{
        num_of_chanels=1;
    }
    dim3 numBlocks(width*num_of_chanels,height);

    hipError_t err = hipSuccess;
    compute_image_kernel<<<width*num_of_chanels, height>>>(xmin, xmax, ymin, ymax, max_iter, width, height, result, num_of_chanels);
    checkErr(err, "Failed to run Kernel\n");
    void *data = malloc(height * width * num_of_chanels * sizeof(char));
    err = hipMemcpy(data, result, width * height * num_of_chanels *sizeof(char), hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy result back\n");

    char name[60];
    if(rgb==0){
	    sprintf(name,"fractal-images/cuda/bw_img%dx%d_%d.jpg", width, height, max_iter);
    }else{
	    sprintf(name,"fractal-images/cuda/rgb_img%dx%d_%d.jpg", width, height, max_iter);
    }
	stbi_write_jpg(name, width, height, num_of_chanels, data, 300);
		
}

int main(int argc, char** argv){
    
	FILE *out_file = fopen("code/data/cuda.txt", "a");
    if (out_file == NULL) {   
		printf("Error! Could not open file\n"); 
        exit(-1);
    } 
    hipError_t err = hipSuccess;

    int width = 480;
    int height = 480;
    int max_iter = 25000;

    int num_of_chanels;
    int rgb = 0;
    double xmin=-1.5;
	double xmax= 0.5;
	double ymin=-1.0;
	double ymax= 1.0;


	if (argc > 1)
    	height = atoi(argv[1]);
	if(argc>2){
		width = atoi(argv[2]);
	}
	if(argc>3){
		max_iter = atoi(argv[3]);
	}
    if(argc>4){
        rgb = atoi(argv[4]);
    }


	clock_t begin = clock();

    if(rgb==0){
        num_of_chanels = 1;
    }
    else{
        num_of_chanels=3;
    }

    char *result = NULL;
    err = hipMalloc(&result, width * height * num_of_chanels * sizeof(char));
    checkErr(err, "Failed to allocate result memory on gpu\n");
    run(xmin, xmax, ymin, ymax, width, height, max_iter, result, rgb);

	clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    
	fprintf(out_file, "%d,%d,%d,%f,%d \n", width, height, max_iter, time_spent,rgb); // write to file 

	printf("time took for execution of cuda parallel algorithm with parameters(%dx%d,%d): %f\n", width,height,max_iter,time_spent);

    hipFree(result);
	return 0;
}